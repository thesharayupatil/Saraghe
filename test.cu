
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#define SIZE 100000
using namespace std;

__global__ void addVect(int *vect1 ,int *vect2 , int *resultVect){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    //printf("Thread id == %d || Block Id == %d\n",threadIdx.x,blockDim.x);
    resultVect[i] = vect1[i] + vect2[i];
}

int main(){
    int *d_inVect1,*d_inVect2,*d_outResultVector;
    int vect1[SIZE],vect2[SIZE];
    int resultVect[SIZE];
    hipEvent_t gpu_start,gpu_stop;
    float gpu_elapsed_time;
                                
    // Initializing both the vectors
    for(int i = 0 ; i < SIZE ; i++){
        vect1[i] = i;
        vect2[i] = i;
    }
    // Parallel code

    // Allocate memory on GPU for 3 vectors
    hipMalloc((void**)&d_inVect1,SIZE*(sizeof(int)));
    hipMalloc((void**)&d_inVect2,SIZE*(sizeof(int)));
    hipMalloc((void**)&d_outResultVector,SIZE*(sizeof(int)));

    // CPY the vector contents
    hipMemcpy(d_inVect1,vect1,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_inVect2,vect2,SIZE*sizeof(int),hipMemcpyHostToDevice);

    // Start record for gpu_start
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);

    int blk = SIZE/1024;
    // Call the kernel
    addVect<<<blk+1,1024>>>(d_inVect1,d_inVect2,d_outResultVector);
    hipDeviceSynchronize();
    hipEventRecord(gpu_stop,0);
    // Copy gpu mem to cpu mem
    hipMemcpy(resultVect,d_outResultVector,SIZE*sizeof(int),hipMemcpyDeviceToHost);
        
    
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time,gpu_start,gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    cout<<"The time taken by GPU is :"<<gpu_elapsed_time<<endl;
    
    // verify that the GPU did the work we requested
    bool success = true;
    int total=0;
    cout<<"\nChecking "<<SIZE<<" values in the array.\n";
    for (int i=0; i<SIZE; i++) {
        if ((vect1[i] + vect2[i]) != resultVect[i]) {
            printf( "Error:  %d + %d != %d\n", vect1[i], vect2[i], resultVect[i] );
            success = false;
        }
        total += 1;
    }
    if (success)  cout<<"We did it "<<total<<"  values correct!\n";

    // Sequential code
    clock_t startTime = clock();
    int resultVect2[SIZE];
    for(int i = 0 ; i < SIZE ; i++){
        resultVect2[i] = vect1[i] * vect2[i];
    }
    clock_t endTime = clock();
    printf("\nTime for sequential: %.4f",((float)(endTime-startTime)/CLOCKS_PER_SEC)*1000);
    printf("\n Speedup= %.4f",(((float)(endTime-startTime)/CLOCKS_PER_SEC)*1000)/gpu_elapsed_time);
    return 0;
}
